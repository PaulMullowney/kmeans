#include "hip/hip_runtime.h"
#include "KmeansCudaKernels.h"

template<class TYPE, const int SHMEMSIZE>
__global__ void _dev_rowTransformMinimum(const int m0, const int m, const int n, 
					 const TYPE * __restrict__ normRowsOfA_squared,
					 const TYPE * __restrict__ normColsOfB_squared,
					 const TYPE * __restrict__ input, 
					 int * __restrict__ output) {

  __shared__ TYPE data[SHMEMSIZE];
  __shared__ int index[SHMEMSIZE];
  __shared__ TYPE rowData;
  TYPE dataReg;
  TYPE indexReg;
  int k = (n+blockDim.x-1)/blockDim.x;

  for (int i=blockIdx.x; i<m; i+=gridDim.x) {
    dataReg = FLT_MAX;
    indexReg = -1;

    if (threadIdx.x==0) rowData = normRowsOfA_squared[i];
    __syncthreads();

    for (int j=0; j<k; ++j) {
      int tid = threadIdx.x + j*blockDim.x;
      if (tid<n) {
        TYPE val = input[i*n+tid];
        val = rowData + normColsOfB_squared[tid] - 2.0*val;
        if (val<dataReg) {
          dataReg = val;
          indexReg = tid;
        }
      }
    }
    data[threadIdx.x] = dataReg;
    index[threadIdx.x] = indexReg;
    __syncthreads();

    /* reduce over the block of 128 threads */
    int shift = SHMEMSIZE >> 1;
    while (shift>=1) {
      if (threadIdx.x<shift) {
        if (data[threadIdx.x+shift]<data[threadIdx.x]) {
          data[threadIdx.x] = data[threadIdx.x+shift];
          index[threadIdx.x] = index[threadIdx.x+shift];
        } else if (data[threadIdx.x+shift]==data[threadIdx.x] &&
		   index[threadIdx.x+shift] < index[threadIdx.x]) {
          data[threadIdx.x] = data[threadIdx.x+shift];
          index[threadIdx.x] = index[threadIdx.x+shift];
	}
      }
      __syncthreads();
      shift >>= 1;
    }
    if (threadIdx.x==0)
      output[m0+i] = index[0];
  }
}


/* Generic Templated interface to calling the CUDA kernel */
template<class TYPE>
DllExport kmeansCudaErrorStatus rowTransformMinimum(const int m0, const int m, const int n, 
						    const TYPE * normRowsOfA_squared, 
						    const TYPE * normColsOfB_squared, 
						    const TYPE * input, int * output) {
  try {
    const int nThreads = 128;
    dim3 grid = dim3(getMaxConcurrentBlocks(), 1, 1);
    dim3 block = dim3(nThreads,1,1);
    CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(_dev_rowTransformMinimum<TYPE),nThreads>, 
					  hipFuncCachePreferShared),ERROR_ROWTRANSFORMMINIMUM);
    CUDA_SAFE_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte),ERROR_ROWTRANSFORMMINIMUM);

    _dev_rowTransformMinimum<TYPE,nThreads><<<grid,block>>>
      (m0,m,n,normRowsOfA_squared,normColsOfB_squared,input,output);
    CUDA_SAFE_CALL(hipGetLastError(),ERROR_ROWTRANSFORMMINIMUM);

  } catch (...) {
    return ERROR_ROWTRANSFORMMINIMUM;
  }
  return NO_ERROR;
}

/* Single precision C entry Point */
kmeansCudaErrorStatus rowTransformMinimumF(const int m0, const int m, const int n, 
					   const float * normRowsOfA_squared, 
					   const float * normColsOfB_squared, 
					   const float * input, int * output) {
  return rowTransformMinimum<float>(m0,m,n,normRowsOfA_squared,normColsOfB_squared,input,output);
}

/* Double precision C entry Point */
kmeansCudaErrorStatus rowTransformMinimumD(const int m0, const int m, const int n,
					   const double * normRowsOfA_squared, 
					   const double * normColsOfB_squared, 
					   const double * input, int * output) {
  return rowTransformMinimum<double>(m0,m,n,normRowsOfA_squared,normColsOfB_squared,input,output);
}
