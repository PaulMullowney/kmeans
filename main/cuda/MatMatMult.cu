#include "hip/hip_runtime.h"
#include "KmeansCudaKernels.h"

#include "multiply.hcu"

template<class TYPE, class VTYPE, const int N_UNROLL, const int DELTA>
__global__ void _dev_MatMatMult(const TYPE * __restrict__ A,
				const TYPE * __restrict__ B,
				TYPE * __restrict__ C) {

  __shared__ VTYPE Ashmem[TILESIZEY][TILESIZEY];
  __shared__ VTYPE Bshmem[TILESIZEY][TILESIZEX];

  int r = blockIdx.y*N_UNROLL*TILESIZE + threadIdx.y;
  int c = blockIdx.x*N_UNROLL*TILESIZE + threadIdx.x;
  VTYPE Creg[N_UNROLL];
  for (int n=0; n<N_UNROLL; ++n)
	zero<VTYPE>(Creg[n]);

  /* matrix matrix multiply */
  TYPE * a = const_cast<TYPE *>(A) + r*dev_nColsA + threadIdx.x;
  TYPE * b = const_cast<TYPE *>(B) + c+threadIdx.y*dev_nColsB;
  multiply<TYPE,VTYPE,N_UNROLL,DELTA>(a, b, Creg, Ashmem, Bshmem);

  /* write the results */
  for (int n=0; n<N_UNROLL; ++n) {
    _dev_writeResults<TYPE,VTYPE>(dev_nRowsA,dev_nColsB,r,c,Creg[n],C);
    r+=TILESIZE;
  }
}



template<class TYPE, class VTYPE, const int N_UNROLL, const int VLENGTH>
__launch_bounds__(64,16)
__global__ void _dev_MatMatMult2(const TYPE * __restrict__ A,
				 const TYPE * __restrict__ B,
				 TYPE * __restrict__ C) {

  __shared__ VTYPE Ashmem[TILESIZE][TILESIZE];
  int r = blockIdx.y*VLENGTH*TILESIZE;
  int c = blockIdx.x*N_UNROLL*TILESIZE;
  int tidx = threadIdx.x + threadIdx.y*blockDim.x;
  
  //VTYPE Creg[TILESIZE] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0};
  VTYPE Creg[TILESIZE];
  for (int i=0; i<TILESIZE; ++i) {
    Creg[i].x=0;
    Creg[i].y=0;
  }
	 
  TYPE * a = const_cast<TYPE *>(A) + r*dev_nColsA + threadIdx.x;
  TYPE * b = const_cast<TYPE *>(B) + c + tidx;

  multiply2<TYPE, VTYPE, N_UNROLL>(tidx, a, b, &(Creg[0]), Ashmem);
  
  /* write the results */
  for (int i=0; i<TILESIZE; ++i) {
    if (r+i<dev_nRowsA && c+tidx<dev_nColsB) {
      C[(r+i)*dev_nColsB + c + tidx] = Creg[i].x;
    }
    if (r+i+TILESIZE<dev_nRowsA && c+tidx<dev_nColsB) {
      C[(r+i+TILESIZE)*dev_nColsB + c + tidx] = Creg[i].y;
    }
  }
}



/* Generic Templated interface to calling the CUDA kernel */
template<class TYPE, class VTYPE,  const int N_UNROLL>
kmeansCudaErrorStatus MatMatMult(const int nRowsA, const int nColsA, const TYPE *A, 
				 const int nColsB, const TYPE *B, TYPE *C) {
  
  try {
#if 1
    const int nBy = (nRowsA+N_UNROLL*TILESIZE-1)/(N_UNROLL*TILESIZE);
    const int nBx = (nColsB+N_UNROLL*TILESIZE-1)/(N_UNROLL*TILESIZE);
    int N = (nColsA+TILESIZE-1)/(TILESIZE);
    int delta = nColsA-(N-1)*TILESIZE;
    N-=1;
    dim3 grid = dim3(nBx, nBy);
    dim3 block = dim3(TILESIZE, TILESIZE);

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int), 0),ERROR_MATMATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nRowsA), &nRowsA, sizeof(int)),ERROR_MATMATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nColsA), &nColsA, sizeof(int)),ERROR_MATMATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nRowsB), &nColsA, sizeof(int)),ERROR_MATMATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nColsB), &nColsB, sizeof(int)),ERROR_MATMATMULT);
    int astride = TILESIZE*nColsA;
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ASTRIDE), &astride, sizeof(int)),ERROR_MATMATMULT);

    CUDA_SAFE_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte),ERROR_MATMATMULT);

    if (delta==1) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,1><<<grid,block>>>(A,B,C);
    if (delta==2) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,2><<<grid,block>>>(A,B,C);
    if (delta==3) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,3><<<grid,block>>>(A,B,C);
    if (delta==4) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,4><<<grid,block>>>(A,B,C);
    if (delta==5) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,5><<<grid,block>>>(A,B,C);
    if (delta==6) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,6><<<grid,block>>>(A,B,C);
    if (delta==7) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,7><<<grid,block>>>(A,B,C);
    if (delta==8) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,8><<<grid,block>>>(A,B,C);
    if (delta==9) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,9><<<grid,block>>>(A,B,C);
    if (delta==10) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,10><<<grid,block>>>(A,B,C);
    if (delta==11) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,11><<<grid,block>>>(A,B,C);
    if (delta==12) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,12><<<grid,block>>>(A,B,C);
    if (delta==13) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,13><<<grid,block>>>(A,B,C);
    if (delta==14) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,14><<<grid,block>>>(A,B,C);
    if (delta==15) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,15><<<grid,block>>>(A,B,C);
    if (delta==16) _dev_MatMatMult<TYPE,VTYPE,N_UNROLL,16><<<grid,block>>>(A,B,C);
#else
    const int nBy = (nRowsA+2*TILESIZE-1)/(2*TILESIZE);
    const int nBx = (nColsB+N_UNROLL*TILESIZE-1)/(N_UNROLL*TILESIZE);
    int N = (nColsA+TILESIZE-1)/(TILESIZE);
    int delta = nColsA-(N-1)*TILESIZE;
    N-=1;

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int), 0),ERROR_MATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nRowsA), &nRowsA, sizeof(int)),ERROR_MATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nColsA), &nColsA, sizeof(int)),ERROR_MATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nRowsB), &nColsA, sizeof(int)),ERROR_MATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nColsB), &nColsB, sizeof(int)),ERROR_MATMULT);
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_DELTA), &delta, sizeof(int)),ERROR_MATMULT);

    dim3 grid = dim3(nBx, nBy);
    dim3 block = dim3(TILESIZE, 4);
    CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(_dev_MatMatMult2<TYPE),float2,4,2>, 
					  hipFuncCachePreferL1),ERROR_MATMATMULT);
    CUDA_SAFE_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte),ERROR_MATMATMULT);
    _dev_MatMatMult2<TYPE,float2,4,2><<<grid,block>>>(A,B,C);
#endif
    CUDA_SAFE_CALL(hipGetLastError(),ERROR_MATMATMULT);
  } catch (...) {
    return ERROR_MATMATMULT;
  }
  return NO_ERROR;
}

/* Single precision C entry Point */
kmeansCudaErrorStatus MatMatMultF(const int nRowsA, const int nColsA, const float *A, 
				  const int nColsB, const float *B, float *C) {
  return MatMatMult<float,FVECTOR,N_UNROLL_FLOAT>(nRowsA,nColsA,A,nColsB,B,C);
}

/* Double precision C entry Point */
kmeansCudaErrorStatus MatMatMultD(const int nRowsA, const int nColsA, const double *A, 
				  const int nColsB, const double *B, double *C) {
  return MatMatMult<double,DVECTOR,N_UNROLL_DOUBLE>(nRowsA,nColsA,A,nColsB,B,C);
}
