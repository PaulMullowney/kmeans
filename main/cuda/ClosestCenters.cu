#include "hip/hip_runtime.h"
#include "KmeansCudaKernels.h"

#include "loadVector.hcu"
#include "reduction.hcu"
#include "multiply.hcu"

template<class TYPE, class VTYPE, const int N_UNROLL, const int DELTA>
//__launch_bounds__(256,1)
__global__ void _dev_ClosestCentersBeginT(const TYPE * __restrict__ A, const TYPE * __restrict__ B,
					  const TYPE * __restrict__ normRowsOfA_squared,
					  const TYPE * __restrict__ normColsOfB_squared,
					  TYPE * __restrict__ C, int * __restrict__ Cindices) {

  __shared__ VTYPE Ashmem[TILESIZEY][TILESIZEX];
  __shared__ VTYPE Bshmem[TILESIZEY][TILESIZEX];

  /* read in the vector data from global memory */
  __shared__ VTYPE L2normB[TILESIZE];

  int r = blockIdx.y*N_UNROLL*TILESIZE + N_UNROLL*threadIdx.x;
  int c = blockIdx.x*N_UNROLL*TILESIZE + N_UNROLL*threadIdx.x;

  VTYPE Creg[N_UNROLL];
  for (int n=0; n<N_UNROLL; ++n)
    zero<VTYPE>(Creg[n]);

  /* matrix matrix multiply */
  TYPE * a = const_cast<TYPE *>(A) + r + threadIdx.y*dev_nRowsAPadded;
  TYPE * b = const_cast<TYPE *>(B) + c + threadIdx.y*dev_nColsB;

  multiplyT<TYPE,VTYPE,N_UNROLL,DELTA>(a, b, Creg, Ashmem, Bshmem);

  /* load the vector data */
  if (threadIdx.y==0)
    _dev_loadVector(c, dev_nColsB, normColsOfB_squared, L2normB[threadIdx.x]);
  __syncthreads();

  /* perform the partial reduction over each row in the shmem buffers */
  _dev_reduction<TILESIZEX,TILESIZEX>(c, Creg, L2normB, Ashmem, Bshmem); 

  a = reinterpret_cast<TYPE *>(&(Ashmem[threadIdx.y][0]));
  b = reinterpret_cast<TYPE *>(&(Bshmem[threadIdx.y][0]));

  /* write out the results */
  if (threadIdx.x<N_UNROLL) {
    int r = blockIdx.y*N_UNROLL*TILESIZE + N_UNROLL*threadIdx.y + threadIdx.x;
    //r += threadIdx.y;
    if(r<dev_nRowsA) {
      C[r*dev_nColsC+blockIdx.x]        = a[threadIdx.x];
      Cindices[r*dev_nColsC+blockIdx.x] = (int)b[threadIdx.x];
    }      
  }
}

template<class TYPE, class VTYPE, const int N_UNROLL, const int DELTA>
//__launch_bounds__(256,1)
__global__ void _dev_ClosestCentersBegin(const TYPE * __restrict__ A, const TYPE * __restrict__ B,
					 const TYPE * __restrict__ normRowsOfA_squared,
					 const TYPE * __restrict__ normColsOfB_squared,
					 TYPE * __restrict__ C, int * __restrict__ Cindices) {

  __shared__ VTYPE Ashmem[TILESIZEY][TILESIZEY];
  __shared__ VTYPE Bshmem[TILESIZEY][TILESIZEX];

  /* read in the vector data from global memory */
  __shared__ VTYPE L2normB[TILESIZE];

  int r = blockIdx.y*N_UNROLL*TILESIZE + N_UNROLL*threadIdx.y;
  int c = blockIdx.x*N_UNROLL*TILESIZE + N_UNROLL*threadIdx.x;

  VTYPE Creg[N_UNROLL];
  for (int n=0; n<N_UNROLL; ++n)
    zero<VTYPE>(Creg[n]);

  /* matrix matrix multiply */
  TYPE * a = const_cast<TYPE *>(A) + r*dev_nColsA + threadIdx.x;
  TYPE * b = const_cast<TYPE *>(B) + c + threadIdx.y*dev_nColsB;

  multiply<TYPE,VTYPE,N_UNROLL,DELTA>(a, b, Creg, Ashmem, Bshmem);

  /* load the vector data */
  if (threadIdx.y==0)
    _dev_loadVector(c, dev_nColsB, normColsOfB_squared, L2normB[threadIdx.x]);
  __syncthreads();

  /* perform the partial reduction over each row in the shmem buffers */
  _dev_reduction<TILESIZEY,TILESIZEX>(c, Creg, L2normB, Ashmem, Bshmem);

  a = reinterpret_cast<TYPE *>(&(Ashmem[threadIdx.y][0]));
  b = reinterpret_cast<TYPE *>(&(Bshmem[threadIdx.y][0]));

  /* write out the results */
  if (threadIdx.x<N_UNROLL) {
    r += threadIdx.x;
    if(r<dev_nRowsA) {
      C[r*dev_nColsC+blockIdx.x]        = a[threadIdx.x];
      Cindices[r*dev_nColsC+blockIdx.x] = (int)b[threadIdx.x];
    }      
  }
}



template<class TYPE, const int N_UNROLL, const int DELTA>
//__launch_bounds__(256,1)
__global__ void _dev_ClosestCentersBeginNew(const TYPE * __restrict__ A, const TYPE * __restrict__ B,
					    const TYPE * __restrict__ normRowsOfA_squared,
					    const TYPE * __restrict__ normColsOfB_squared,
					    TYPE * __restrict__ C, int * __restrict__ Cindices) {

  __shared__ TYPE Ashmem[N_UNROLL*TILESIZEY][TILESIZEY];
  __shared__ TYPE Bshmem[N_UNROLL*TILESIZEY][TILESIZEX];

  /* read in the vector data from global memory */
  __shared__ TYPE L2normB[N_UNROLL*TILESIZE];

  int r = blockIdx.y*N_UNROLL*TILESIZE + N_UNROLL*threadIdx.y;
  int c = blockIdx.x*N_UNROLL*TILESIZE + N_UNROLL*threadIdx.x;

  TYPE Creg[N_UNROLL*N_UNROLL];
  for (int n=0; n<N_UNROLL*N_UNROLL; ++n)
    Creg[n]=0;

  /* matrix matrix multiply */
  TYPE * a = const_cast<TYPE *>(A) + r*dev_nColsA + threadIdx.x;
  TYPE * b = const_cast<TYPE *>(B) + c + threadIdx.y*dev_nColsB;

  multiplyNew<TYPE,N_UNROLL,DELTA>(a, b, Creg, Ashmem, Bshmem);

  /* load the vector data */
  if (threadIdx.y==0) {
    for (int n=0; n<N_UNROLL; ++n) {
      L2normB[threadIdx.x*N_UNROLL+n] = 0;
      if (c+n<dev_nColsB)
	L2normB[threadIdx.x*N_UNROLL+n] = normColsOfB_squared[c+n];
    }
  }
  __syncthreads();

#if 0

  /* perform the partial reduction over each row in the shmem buffers */
  _dev_reduction<TYPE,TILESIZEY,TILESIZEX,N_UNROLL>(c, Creg, L2normB, Ashmem, Bshmem);

  a = reinterpret_cast<TYPE *>(&(Ashmem[threadIdx.y][0]));
  b = reinterpret_cast<TYPE *>(&(Bshmem[threadIdx.y][0]));

  /* write out the results */
  if (threadIdx.x<N_UNROLL) {
    r += threadIdx.x;
    if(r<dev_nRowsA) {
      C[r*dev_nColsC+blockIdx.x]        = a[threadIdx.x];
      Cindices[r*dev_nColsC+blockIdx.x] = (int)b[threadIdx.x];
    }      
  }
#endif
}

__host__ __device__ static __inline__ int nextPowerOfTwo(int v) {
  v--;
  v |= v >> 1;
  v |= v >> 2;
  v |= v >> 4;
  v |= v >> 8;
  v |= v >> 16;
  v++;
  return v;
}

template<class TYPE, const int SHMEMSIZE>
__global__ void _dev_ClosestCentersEnd(const int m0, const int n, const int k, const int SHIFT,
				       const TYPE * __restrict__ C,
				       const int * __restrict__ Cindices,
				       int * __restrict__ CindicesFinal) {

  __shared__ TYPE data[SHMEMSIZE];
  __shared__ int index[SHMEMSIZE];
  TYPE dataReg;
  TYPE indexReg;

  for (int i=blockIdx.x; i<n; i+=gridDim.x) {
    dataReg = FLT_MAX;
    indexReg = -1;

    for (int j=threadIdx.x; j<k; j+=blockDim.x) {
      TYPE val = C[i*k+j];
      int index = Cindices[i*k+j];
      if (val<dataReg) {
	dataReg = val;
	indexReg = index;
      }
    }
    data[threadIdx.x] = dataReg;
    index[threadIdx.x] = indexReg;
    __syncthreads();

    /* reduce over the block of 128 threads */
    int shift = SHIFT;
    int j;
    while (shift>=1) {
      if (threadIdx.x<shift) {
	data[threadIdx.x] = fminf(data[threadIdx.x],data[threadIdx.x+shift],
				  index[threadIdx.x],index[threadIdx.x+shift], j);
	index[threadIdx.x] = j;
      }
      __syncthreads();
      shift >>= 1;
    }
    if (threadIdx.x==0)
      CindicesFinal[m0 + i] = index[0];
    __syncthreads();
  }  
}

/* Generic Templated interface to calling the CUDA kernel */
template<class TYPE, class VTYPE,  const int N_UNROLL>
kmeansCudaErrorStatus ClosestCenters(const int m0, const int nRowsA, const int nColsA,
				     const bool isTranspose, const TYPE *A, 
				     const int nColsB, const TYPE *B, 
				     const TYPE * normRowsOfA_squared,
				     const TYPE * normColsOfB_squared,
				     const int nColsC, TYPE * C, int *Cindices,
				     int * CindicesFinal, bool& constantMemSet) {
  
  try {
    const int nBy = (nRowsA+N_UNROLL*TILESIZE-1)/(N_UNROLL*TILESIZE);
    const int nBx = (nColsB+N_UNROLL*TILESIZE-1)/(N_UNROLL*TILESIZE);
    int N = (nColsA+TILESIZE-1)/(TILESIZE);
    int delta = nColsA-(N-1)*TILESIZE;
    N-=1;
    dim3 grid = dim3(nBx, nBy);
    dim3 block = dim3(TILESIZE, TILESIZE);


    if (constantMemSet==false) {
      int nRowsAPadded = ((nRowsA + N_UNROLL*TILESIZE-1)/(N_UNROLL*TILESIZE))*N_UNROLL*TILESIZE;
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_N), &N, sizeof(int), 0),ERROR_CLOSESTCENTERS);
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nRowsA), &nRowsA, sizeof(int)),ERROR_CLOSESTCENTERS);
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nRowsAPadded), &nRowsAPadded, sizeof(int)),ERROR_CLOSESTCENTERS);
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nColsA), &nColsA, sizeof(int)),ERROR_CLOSESTCENTERS);
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nRowsB), &nColsA, sizeof(int)),ERROR_CLOSESTCENTERS);
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nColsB), &nColsB, sizeof(int)),ERROR_CLOSESTCENTERS);
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dev_nColsC), &nColsC, sizeof(int)),ERROR_CLOSESTCENTERS);
      //int astride = TILESIZE*nColsA;
      int astride = nColsA;
      CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(ASTRIDE), &astride, sizeof(int)),ERROR_CLOSESTCENTERS);
      constantMemSet = true;
    }
    CUDA_SAFE_CALL(hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte),ERROR_CLOSESTCENTERS);
    
    if (isTranspose) {

      if (delta==1)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,1><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==2)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,2><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==3)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,3><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==4)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,4><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==5)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,5><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==6)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,6><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==7)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,7><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==8)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,8><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==9)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,9><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==10)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,10><<<grid,block>>>(A,B,normRowsOfA_squared,
									  normColsOfB_squared,C,Cindices);
      else if (delta==11)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,11><<<grid,block>>>(A,B,normRowsOfA_squared,
									  normColsOfB_squared,C,Cindices);
      else if (delta==12)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,12><<<grid,block>>>(A,B,normRowsOfA_squared,
									  normColsOfB_squared,C,Cindices);
      else if (delta==13)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,13><<<grid,block>>>(A,B,normRowsOfA_squared,
									  normColsOfB_squared,C,Cindices);
      else if (delta==14)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,14><<<grid,block>>>(A,B,normRowsOfA_squared,
									  normColsOfB_squared,C,Cindices);
      else if (delta==15)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,15><<<grid,block>>>(A,B,normRowsOfA_squared,
									  normColsOfB_squared,C,Cindices);
      else if (delta==16)
	_dev_ClosestCentersBeginT<TYPE,VTYPE,N_UNROLL,16><<<grid,block>>>(A,B,normRowsOfA_squared,
									  normColsOfB_squared,C,Cindices);
      
    } else {

      if (delta==1)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,1><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==2)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,2><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==3)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,3><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==4)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,4><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==5)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,5><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==6)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,6><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==7)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,7><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==8)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,8><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==9)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,9><<<grid,block>>>(A,B,normRowsOfA_squared,
									normColsOfB_squared,C,Cindices);
      else if (delta==10)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,10><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==11)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,11><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==12)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,12><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==13)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,13><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==14)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,14><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==15)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,15><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
      else if (delta==16)
	_dev_ClosestCentersBeginNew<TYPE,N_UNROLL,16><<<grid,block>>>(A,B,normRowsOfA_squared,
									 normColsOfB_squared,C,Cindices);
    }

    CUDA_SAFE_CALL(hipGetLastError(),ERROR_CLOSESTCENTERS);

    const int nThreads = 128;
    grid = dim3(getMaxConcurrentBlocks(), 1, 1);
    block = dim3(nThreads,1,1);

    int shift = nextPowerOfTwo(nColsC);
    //CUDA_SAFE_CALL(hipFuncSetCacheConfig(reinterpret_cast<const void*>(_dev_ClosestCentersEnd<TYPE),nThreads>,
    //					  hipFuncCachePreferShared),ERROR_CLOSESTCENTERS);
    _dev_ClosestCentersEnd<TYPE,nThreads><<<grid,block>>>(m0,nRowsA,nColsC,shift,C,Cindices,CindicesFinal);
    
    CUDA_SAFE_CALL(hipGetLastError(),ERROR_CLOSESTCENTERS);
  } catch (...) {
    return ERROR_CLOSESTCENTERS;
  }
  return NO_ERROR;
}


/* Generic Templated interface to calling the CUDA kernel */
template<class TYPE>
DllExport kmeansCudaErrorStatus ClosestCenters(const int m0, const int nRowsA, const int nColsA,
					       const bool isTranspose, const TYPE *A,
					       const int nColsB, const TYPE *B, 
					       const TYPE * normRowsOfA_squared,
					       const TYPE * normColsOfB_squared,
					       const int nColsC, TYPE * C, int *Cindices,
					       int * CindicesFinal, bool& constantMemSet) {
  return NO_ERROR;
}

template<>
kmeansCudaErrorStatus ClosestCenters(const int m0, const int nRowsA, const int nColsA,
				     const bool isTranspose, const float *A,
				     const int nColsB, const float *B, 
				     const float * normRowsOfA_squared,
				     const float * normColsOfB_squared,
				     const int nColsC, float * C, int *Cindices,
				     int * CindicesFinal, bool& constantMemSet) {

  return ClosestCenters<float,FVECTOR,N_UNROLL_FLOAT>
    (m0,nRowsA,nColsA,isTranspose,(const float *)A,nColsB,(const float *)B,
     (const float *)normRowsOfA_squared, (const float *)normColsOfB_squared,
     nColsC,(float *)C,Cindices,CindicesFinal, constantMemSet);
}


template<>
kmeansCudaErrorStatus ClosestCenters(const int m0, const int nRowsA, const int nColsA,
				     const bool isTranspose, const double *A,
				     const int nColsB, const double *B, 
				     const double * normRowsOfA_squared,
				     const double * normColsOfB_squared,
				     const int nColsC, double * C, int *Cindices,
				     int * CindicesFinal, bool& constantMemSet) {
  
  return ClosestCenters<double,DVECTOR,N_UNROLL_DOUBLE>
    (m0,nRowsA,nColsA,isTranspose,(const double *)A,nColsB,(const double *)B,
     (const double *)normRowsOfA_squared, (const double *)normColsOfB_squared,
     nColsC,(double *)C,Cindices,CindicesFinal, constantMemSet);
}



/* Single precision C entry Point */
kmeansCudaErrorStatus ClosestCentersF(const int m0, const int nRowsA, const int nColsA,
				      const float *A, const int nColsB, const float *B, 
				      const float * normRowsOfA_squared,
				      const float * normColsOfB_squared,
				      const int nColsC, float * C, int *Cindices,
				      int * CindicesFinal) {
  
  bool constantMemSet = false;
  return ClosestCenters<float,FVECTOR,N_UNROLL_FLOAT>
    (m0,nRowsA,nColsA,false,A,nColsB,B,normRowsOfA_squared,
     normColsOfB_squared,nColsC,C,Cindices,CindicesFinal, constantMemSet);
}

/* Double precision C entry Point */
kmeansCudaErrorStatus ClosestCentersD(const int m0, const int nRowsA, const int nColsA, 
				      const double *A, const int nColsB, const double *B, 
				      const double * normRowsOfA_squared,
				      const double * normColsOfB_squared,
				      const int nColsC, double * C, int *Cindices,
				      int * CindicesFinal) {

  bool constantMemSet = false;
  return ClosestCenters<double,DVECTOR,N_UNROLL_DOUBLE>
    (m0,nRowsA,nColsA,false,A,nColsB,B,normRowsOfA_squared,
     normColsOfB_squared,nColsC,C,Cindices,CindicesFinal, constantMemSet);
}
